#include "hip/hip_runtime.h"
#include "cProteinLoader.h"
#include <iostream>
#include <string>
#include <THC/THC.h>
#include <math.h>

#define d 2
	__global__ void projectToTensor(float* d_flat_data, int* d_n_atoms, size_t* d_offsets, float *out, 
									int batch_size, int num_atom_types, int spatial_dim,
									float res){
		size_t func_index = threadIdx.x + blockIdx.x*blockDim.x;
		float *volume = out + func_index * spatial_dim*spatial_dim*spatial_dim;
		float *atoms_coords = d_flat_data + d_offsets[func_index];
		int n_atoms = d_n_atoms[func_index];
		for(int atom_idx = 0; atom_idx<n_atoms; atom_idx+=3){
			float 	x = atoms_coords[atom_idx],
					y = atoms_coords[atom_idx + 1],
					z = atoms_coords[atom_idx + 2];
			int x_i = floor(x/res);
			int y_i = floor(y/res);
			int z_i = floor(z/res);
			for(int i=x_i-d; i<=(x_i+d);i++){
				for(int j=y_i-d; j<=(y_i+d);j++){
					for(int k=z_i-d; k<=(z_i+d);k++){
						if( (i>=0 && i<spatial_dim) && (j>=0 && j<spatial_dim) && (k>=0 && k<spatial_dim) ){
							int idx = k + j*spatial_dim + i*spatial_dim*spatial_dim;							
							float r2 = (x - i*res)*(x - i*res)+\
							(y - j*res)*(y - j*res)+\
							(z - k*res)*(z - k*res);
							volume[idx]+=exp(-r2/2.0);
						}
					}
				}
			}
		}
	}

extern "C"{
	int loadProtein(const char* proteinPath, 
					bool shift, bool rot, float resolution, 
					int assigner_type, int spatial_dim, THGenerator *gen, 
					float** data_pointer, int* n_atoms){

		cProteinLoader pL;
		pL.loadPDB(proteinPath);
		if(pL.assignAtomTypes(assigner_type)<0){
			return -1;
		}
		
		pL.res = resolution;
		pL.computeBoundingBox();
		//placing center of the bbox to the origin
		pL.shiftProtein( -0.5*(pL.b0 + pL.b1) ); 
		if(rot){
			// float alpha = THRandom_uniform(gen,0,2.0*M_PI);
 			// float beta = THRandom_uniform(gen,0,2.0*M_PI);
 			// float theta = THRandom_uniform(gen,0,2.0*M_PI);
 			// cMatrix33 random_rotation = cMatrix33::rotationXYZ(alpha,beta,theta);
			float u1 = THRandom_uniform(gen,0,1.0);
			float u2 = THRandom_uniform(gen,0,1.0);
			float u3 = THRandom_uniform(gen,0,1.0);
			float q[4];
			q[0] = sqrt(1-u1) * sin(2.0*M_PI*u2);
			q[1] = sqrt(1-u1) * cos(2.0*M_PI*u2);
			q[2] = sqrt(u1) * sin(2.0*M_PI*u3);
			q[3] = sqrt(u1) * cos(2.0*M_PI*u3);
			cMatrix33 random_rotation;
			random_rotation.m[0][0] = q[0]*q[0] + q[1]*q[1] - q[2]*q[2] - q[3]*q[3];
			random_rotation.m[0][1] = 2.0*(q[1]*q[2] - q[0]*q[3]);
			random_rotation.m[0][2] = 2.0*(q[1]*q[3] + q[0]*q[2]);

			random_rotation.m[1][0] = 2.0*(q[1]*q[2] + q[0]*q[3]);
			random_rotation.m[1][1] = q[0]*q[0] - q[1]*q[1] + q[2]*q[2] - q[3]*q[3];
			random_rotation.m[1][2] = 2.0*(q[2]*q[3] - q[0]*q[1]);

			random_rotation.m[2][0] = 2.0*(q[1]*q[3] - q[0]*q[2]);
			random_rotation.m[2][1] = 2.0*(q[2]*q[3] + q[0]*q[1]);
			random_rotation.m[2][2] = q[0]*q[0] - q[1]*q[1] - q[2]*q[2] + q[3]*q[3];
			pL.rotateProtein(random_rotation);
		}
		if(shift){
			float dx_max = fmax(0, spatial_dim*pL.res/2.0 - (pL.b1[0]-pL.b0[0])/2.0)*0.5;
			float dy_max = fmax(0, spatial_dim*pL.res/2.0 - (pL.b1[1]-pL.b0[1])/2.0)*0.5;
			float dz_max = fmax(0, spatial_dim*pL.res/2.0 - (pL.b1[2]-pL.b0[2])/2.0)*0.5;
			float dx = THRandom_uniform(gen,-dx_max,dx_max);
		 	float dy = THRandom_uniform(gen,-dy_max,dy_max);
		 	float dz = THRandom_uniform(gen,-dz_max,dz_max);
		 	pL.shiftProtein(cVector3(dx,dy,dz));
		}
		// placing center of the protein to the center of the grid
		pL.shiftProtein( 0.5*cVector3(spatial_dim, spatial_dim, spatial_dim)*pL.res ); 

		for(int i=0; i<pL.num_atom_types; i++){
			std::vector<float> coords; // vector of plain coords of a particular atom type
			for(int j=0; j<pL.atomType.size();j++){
				if(pL.atomType[j]==i){
					coords.push_back(pL.r[j].v[0]);
					coords.push_back(pL.r[j].v[1]);
					coords.push_back(pL.r[j].v[2]);
				}
			}
			float *coords_plain = new float[coords.size()];
			data_pointer[i] = coords_plain;
			n_atoms[i] = coords.size();
			for(int j=0; j<coords.size();j++)
				coords_plain[j]=coords[j];
		}
		return 1;
	}

	typedef struct{
		char **strings;
		size_t len;
		size_t ind;
	} batchInfo;

	batchInfo* createBatchInfo(int batch_size){
		//std::cout<<"Creating batch info of size = "<<batch_size<<"\n";
		batchInfo *binfo;
		binfo = new batchInfo;
		binfo->strings = new char*[batch_size];
		binfo->len = batch_size;
		binfo->ind = 0;
		return binfo;
	}

	void deleteBatchInfo(batchInfo* binfo){
		for(int i=0;i<binfo->len;i++){
			delete [] binfo->strings[i];
		}
		binfo->len=0;
		binfo->ind=0;
		delete binfo;
	}

	void pushProteinToBatchInfo(const char* filename, batchInfo* binfo){
		std::string str(filename);
		//std::cout<<"Pushing the string "<<str<<" to the position "<<pos<<"\n";
		//std::cout<<grid4D->nDimension<<"\n";

		binfo->strings[binfo->ind] = new char[str.length()+1];
		for(int i=0; i<str.length(); i++){
			binfo->strings[binfo->ind][i] = str[i];
		}
		binfo->strings[binfo->ind][str.length()]='\0';
		binfo->ind += 1;
		//std::cout<<binfo->grids4D[pos]->nDimension<<"\n";
	}

	void printBatchInfo(batchInfo* binfo){
		for(int i=0;i<min(binfo->len,binfo->ind);i++){
			std::cout<<binfo->strings[i]<<"\n";
		}
	}
	

	int loadProteinCUDA(THCState *state,
						 batchInfo* batch, THCudaTensor *batch5D,
						 bool shift, bool rot, float resolution,
						 int assigner_type, int spatial_dim){
		// std::cout<<"Flag!"<<std::endl;
		THGenerator *gen = THGenerator_new();
 		THRandom_seed(gen);
 		int num_atom_types;
 		if(assigner_type==1)num_atom_types=4;
 		else num_atom_types=11;
		// std::cout<<"Launched function"<<(batch->len)*num_atom_types<<std::endl;
 		float **data_array = new float*[(batch->len)*num_atom_types];
 		int *n_atoms = new int[(batch->len)*num_atom_types];
 		size_t *offsets = new size_t[(batch->len)*num_atom_types];
 		
 		std::vector<int> flags;
 		flags.resize(batch->len);
		#pragma omp parallel for num_threads(10)
		for(int i=0; i<batch->len; i++){
			int res = loadProtein(batch->strings[i], shift, rot, resolution, assigner_type, spatial_dim, gen, 
				data_array + i*num_atom_types, n_atoms + i*num_atom_types);
			flags[i] = res;
		}
		
		for(int i=0; i<batch->len; i++){
			if(flags[i]<0){
				std::cout<<"Corrupt file detected\n";
				for(int j=0;j<batch->len;j++){
					if(flags[j]>0)
						for(int k=0;k<num_atom_types;k++)
							delete[] data_array[k+j*num_atom_types];	
				}
				delete[] data_array;
				delete[] n_atoms;
				delete[] offsets;
				THGenerator_free(gen);
				return -1;
			}
		}

		size_t total_size = 0;
		for(int batch_idx=0; batch_idx<batch->len; batch_idx++){
			for(int a_type_idx=0; a_type_idx<num_atom_types; a_type_idx++){
				int volume_idx = a_type_idx + batch_idx*num_atom_types;
				if(volume_idx>0)
					offsets[volume_idx] = offsets[volume_idx - 1] + n_atoms[volume_idx-1];
				else
					offsets[volume_idx] = 0;
				total_size += n_atoms[volume_idx];
			}
		}

		// for(int i=0;i<batch->len;i++){
		// 	for(int j=0;j<num_atom_types;j++){
		// 		std::cout<<"("<<n_atoms[j+i*num_atom_types]<<","<<offsets[j+i*num_atom_types]<<"), ";
		// 	}
		// 	std::cout<<"\n";
		// }

		// for(int i=0;i<n_atoms[1];i++){
		// 	std::cout<<data_array[1][i]<<"\n";
		// }

		//data_array copy to gpu
		float *d_flat_data;
		int *d_n_atoms;
		size_t *d_offsets;
		hipMalloc( (void**) &d_flat_data, total_size*sizeof(float) );
		hipMalloc( (void**) &d_n_atoms, (batch->len)*num_atom_types*sizeof(int) );
		hipMalloc( (void**) &d_offsets, (batch->len)*num_atom_types*sizeof(size_t) );
				
		hipMemcpy( d_n_atoms, n_atoms, (batch->len)*num_atom_types*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy( d_offsets, offsets, (batch->len)*num_atom_types*sizeof(size_t), hipMemcpyHostToDevice);
		
		for(int i=0; i<batch->len; i++){
			for(int j=0; j<num_atom_types; j++){
				int volume_idx = j+i*num_atom_types;
				hipMemcpy( d_flat_data + offsets[volume_idx], data_array[j+i*num_atom_types], 
							n_atoms[j+i*num_atom_types]*sizeof(float), hipMemcpyHostToDevice);
			}
		}
		float* grid = THCudaTensor_data(state, batch5D);
		projectToTensor<<<(batch->len), num_atom_types>>>(	d_flat_data, d_n_atoms, d_offsets,
															grid, 
															batch->len, num_atom_types, spatial_dim,
															resolution);
	
				
		for(int i=0;i<batch->len*num_atom_types;i++){
			delete[] data_array[i];
		}
		delete[] data_array;
		delete[] n_atoms;
		delete[] offsets;
		hipFree(d_n_atoms);
		hipFree(d_flat_data);
		hipFree(d_offsets);
		THGenerator_free(gen);
		return 1;
	}
}