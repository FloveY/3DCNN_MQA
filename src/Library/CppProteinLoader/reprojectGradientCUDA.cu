#include "hip/hip_runtime.h"
#include "cProteinLoader.h"
#include <iostream>
#include <string>
#include <THC/THC.h>
#include <math.h>

#define d 2
	__global__ void projectTensorToAtoms(float* d_flat_data, int* d_n_atoms, size_t* d_offsets, float *grad,
                                        int batch_size, int num_atom_types, int spatial_dim,
									    float res)
    /*
    Input:
        d_flat_data: coordinates in a flat array:
            flat_data: {protein1, ... proteinN}
            protein1: {atom_type1 .. atom_typeM}
            atom_type: {x1,y1,z1 .. xL,yL,zL}
        d_n_atoms: number of atoms in each atom_type 
        d_offsets: offset for coordinates for each atom_type volume
        grad: gradient to be projected on atoms
    Output: 
        d_flat_data: coordinates are rewritten for each atom to store the gradient projection
    */
    {
		size_t func_index = threadIdx.x + blockIdx.x*blockDim.x;
		float *volume = grad + func_index * spatial_dim*spatial_dim*spatial_dim;
		float *atoms_coords = d_flat_data + d_offsets[func_index];
		int n_atoms = d_n_atoms[func_index];
		for(int atom_idx = 0; atom_idx<n_atoms; atom_idx+=3){
			float 	x = atoms_coords[atom_idx],
					y = atoms_coords[atom_idx + 1],
					z = atoms_coords[atom_idx + 2];
            atoms_coords[atom_idx] = 0.0;
            atoms_coords[atom_idx+1] = 0.0;
            atoms_coords[atom_idx+2] = 0.0;
			int x_i = floor(x/res);
			int y_i = floor(y/res);
			int z_i = floor(z/res);
			for(int i=x_i-d; i<=(x_i+d);i++){
				for(int j=y_i-d; j<=(y_i+d);j++){
					for(int k=z_i-d; k<=(z_i+d);k++){
						if( (i>=0 && i<spatial_dim) && (j>=0 && j<spatial_dim) && (k>=0 && k<spatial_dim) ){
							int idx = k + j*spatial_dim + i*spatial_dim*spatial_dim;							
							float r2 = (x - i*res)*(x - i*res)+\
							(y - j*res)*(y - j*res)+\
							(z - k*res)*(z - k*res);
							atoms_coords[atom_idx] -= i*res*volume[idx]*exp(-r2/2.0);
                            atoms_coords[atom_idx + 1] -= j*res*volume[idx]*exp(-r2/2.0);
                            atoms_coords[atom_idx + 2] -= k*res*volume[idx]*exp(-r2/2.0);
						}
					}
				}
			}
		}
	}

extern "C"{
	int prepareProtein(const char* proteinPath, 
					float resolution, 
					int assigner_type, int spatial_dim, 
					float** data_pointer, int* n_atoms){
        /*
        Prepares protein for projecting on the grid.
        */
		cProteinLoader pL;
		pL.loadPDB(proteinPath);
		if(pL.assignAtomTypes(assigner_type)<0){
			return -1;
		}
		
		pL.res = resolution;
		pL.computeBoundingBox();
		//placing center of the bbox to the origin
		pL.shiftProtein( -0.5*(pL.b0 + pL.b1) ); 
		// placing center of the protein to the center of the grid
		pL.shiftProtein( 0.5*cVector3(spatial_dim, spatial_dim, spatial_dim)*pL.res ); 

		for(int i=0; i<pL.num_atom_types; i++){
			std::vector<float> coords; // vector of plain coords of a particular atom type
			for(int j=0; j<pL.atomType.size();j++){
				if(pL.atomType[j]==i){
					coords.push_back(pL.r[j].v[0]);
					coords.push_back(pL.r[j].v[1]);
					coords.push_back(pL.r[j].v[2]);
				}
			}
			float *coords_plain = new float[coords.size()];
			data_pointer[i] = coords_plain;
			n_atoms[i] = coords.size();
			for(int j=0; j<coords.size();j++)
				coords_plain[j]=coords[j];
		}
		return 1;
	}

    int saveProtein(const char* proteinPath, 
					int assigner_type,
					float** data_pointer, int* n_atoms){

		cProteinLoader pL;
		pL.loadPDB(proteinPath);
		if(pL.assignAtomTypes(assigner_type)<0){
			return -1;
		}
        pL.dr.resize(pL.r.size());
		std::cout<<"Assigned atom types"<<std::endl;
		for(int i=0; i<pL.num_atom_types; i++){
            std::vector<float> indexes; // vector of atom indexes for plain coords
			for(int j=0; j<pL.atomType.size(); j++){
				if(pL.atomType[j]==i){
					indexes.push_back(j);
					indexes.push_back(j);
					indexes.push_back(j);
				}
			}
            for(int j=0; j<n_atoms[i]; j+=3){
                cVector3 dr(data_pointer[i][j], data_pointer[i][j+1], data_pointer[i][j+2]);
                pL.dr[indexes[j]] = dr;
            }
		}
        pL.savePDB(proteinPath);
		return 1;
	}

	typedef struct{
		char **strings;
		size_t len;
		size_t ind;
	} batchInfo;

	batchInfo* createBatchInfo(int batch_size){
		//std::cout<<"Creating batch info of size = "<<batch_size<<"\n";
		batchInfo *binfo;
		binfo = new batchInfo;
		binfo->strings = new char*[batch_size];
		binfo->len = batch_size;
		binfo->ind = 0;
		return binfo;
	}

	void deleteBatchInfo(batchInfo* binfo){
		for(int i=0;i<binfo->len;i++){
			delete [] binfo->strings[i];
		}
		binfo->len=0;
		binfo->ind=0;
		delete binfo;
	}

	void pushProteinToBatchInfo(const char* filename, batchInfo* binfo){
		std::string str(filename);
		//std::cout<<"Pushing the string "<<str<<" to the position "<<pos<<"\n";
		//std::cout<<grid4D->nDimension<<"\n";

		binfo->strings[binfo->ind] = new char[str.length()+1];
		for(int i=0; i<str.length(); i++){
			binfo->strings[binfo->ind][i] = str[i];
		}
		binfo->strings[binfo->ind][str.length()]='\0';
		binfo->ind += 1;
		//std::cout<<binfo->grids4D[pos]->nDimension<<"\n";
	}

	void printBatchInfo(batchInfo* binfo){
		for(int i=0;i<min(binfo->len,binfo->ind);i++){
			std::cout<<binfo->strings[i]<<"\n";
		}
	}
	

	int getGradientsCUDA(THCState *state,
						 batchInfo* batch, THCudaTensor *batch5D,
						 float resolution,
						 int assigner_type, int spatial_dim){
 		int num_atom_types;
 		if(assigner_type==1)num_atom_types=4;
 		else num_atom_types=11;
		// std::cout<<"Launched function"<<(batch->len)*num_atom_types<<std::endl;
 		float **data_array = new float*[(batch->len)*num_atom_types];
 		int *n_atoms = new int[(batch->len)*num_atom_types];
 		size_t *offsets = new size_t[(batch->len)*num_atom_types];
 		
 		std::vector<int> flags;
 		flags.resize(batch->len);
		#pragma omp parallel for num_threads(10)
		for(int i=0; i<batch->len; i++){
			int res = prepareProtein(   batch->strings[i], resolution, assigner_type, spatial_dim, 
				                        data_array + i*num_atom_types, n_atoms + i*num_atom_types);
			flags[i] = res;
		}

		for(int i=0; i<batch->len; i++){
			if(flags[i]<0){
				std::cout<<"Corrupt file detected\n";
				for(int j=0;j<batch->len;j++){
					if(flags[j]>0)
						for(int k=0;k<num_atom_types;k++)
							delete[] data_array[k+j*num_atom_types];	
				}
				delete[] data_array;
				delete[] n_atoms;
				delete[] offsets;
				return -1;
			}
		}

		size_t total_size = 0;
		for(int batch_idx=0; batch_idx<batch->len; batch_idx++){
			for(int a_type_idx=0; a_type_idx<num_atom_types; a_type_idx++){
				int volume_idx = a_type_idx + batch_idx*num_atom_types;
				if(volume_idx>0)
					offsets[volume_idx] = offsets[volume_idx - 1] + n_atoms[volume_idx-1];
				else
					offsets[volume_idx] = 0;
				total_size += n_atoms[volume_idx];
			}
		}

		//data_array copy to gpu
		float *d_flat_data;
		int *d_n_atoms;
		size_t *d_offsets;
		hipMalloc( (void**) &d_flat_data, total_size*sizeof(float) );
		hipMalloc( (void**) &d_n_atoms, (batch->len)*num_atom_types*sizeof(int) );
		hipMalloc( (void**) &d_offsets, (batch->len)*num_atom_types*sizeof(size_t) );
				
		hipMemcpy( d_n_atoms, n_atoms, (batch->len)*num_atom_types*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy( d_offsets, offsets, (batch->len)*num_atom_types*sizeof(size_t), hipMemcpyHostToDevice);
		
		for(int i=0; i<batch->len; i++){
			for(int j=0; j<num_atom_types; j++){
				int volume_idx = j+i*num_atom_types;
				hipMemcpy( d_flat_data + offsets[volume_idx], data_array[j+i*num_atom_types], 
							n_atoms[j+i*num_atom_types]*sizeof(float), hipMemcpyHostToDevice);
			}
		}
		float* grid = THCudaTensor_data(state, batch5D);
		std::cout<<"Start project gradient"<<std::endl;
		projectTensorToAtoms<<<(batch->len), num_atom_types>>>(	d_flat_data, d_n_atoms, d_offsets,
															    grid, batch->len, num_atom_types, spatial_dim,
															    resolution);
		std::cout<<"Projected gradient"<<std::endl;
        for(int i=0; i<batch->len; i++){
			for(int j=0; j<num_atom_types; j++){
				int volume_idx = j+i*num_atom_types;
				hipMemcpy( data_array[j+i*num_atom_types], d_flat_data + offsets[volume_idx],
							n_atoms[j+i*num_atom_types]*sizeof(float), hipMemcpyDeviceToHost);
			}
		}
		std::cout<<"Copied memory back"<<std::endl;
        // #pragma omp parallel for num_threads(10)
		for(int i=0; i<batch->len; i++){
			saveProtein(batch->strings[i], assigner_type,
				        data_array + i*num_atom_types, n_atoms + i*num_atom_types);			
		}
		std::cout<<"Saved protein"<<std::endl;
		for(int i=0;i<batch->len*num_atom_types;i++){
			delete[] data_array[i];
		}
		delete[] data_array;
		delete[] n_atoms;
		delete[] offsets;
		hipFree(d_n_atoms);
		hipFree(d_flat_data);
		hipFree(d_offsets);
		return 1;
	}
}